
#include <hip/hip_runtime.h>
__global__ void dither(const float* input,float *error,float *output, int m, int n, int iter,int offset){
	float value=0.0f;
	float e=0.0f;
	float eAux=0.0f;
	int label=0;
	for(int globalTidX=threadIdx.x+blockIdx.x*blockDim.x;globalTidX<offset;globalTidX+=blockDim.x*gridDim.x){
		value=input[iter*offset+globalTidX];
		e=0.0f;
		label=iter+1;
		if (!isnan(value)){
			if(label<=n){
				if((iter-1)>-1){
					eAux=error[(iter-1)*offset+globalTidX];
					(!isnan(eAux))?e+=eAux*0.4375:e+=0.0f;		
					if((globalTidX-1)>-1){
						eAux=error[(iter-1)*offset+(globalTidX-1)];
						(!isnan(eAux))?e+=eAux*0.1875:e+=0.0f;		
					}
				}
				if((iter-2)>-1){
					if((globalTidX-1)>-1){
						eAux=error[(iter-2)*offset+(globalTidX-1)];
						(!isnan(eAux))?e+=eAux*0.3125:e+=0.0f;		
					}
				}
				if((iter-3)>-1){
					if((globalTidX-1)>-1){
						eAux=error[(iter-3)*offset+(globalTidX-1)];
						(!isnan(eAux))?e+=eAux*0.0625:e+=0.0f;		
					}
				}
			}
			else{
				if((n-label)%2){
					if((iter-1)>-1){
						eAux=error[(iter-1)*offset+globalTidX];
						(!isnan(eAux))?e+=eAux*0.1875:e+=0.0f;		
						if((globalTidX+1)<offset){
							eAux=error[(iter-1)*offset+(globalTidX+1)];
							(!isnan(eAux))?e+=eAux*0.4375:e+=0.0f;		
						}
					}
				}
				else{
					if((iter-1)>-1){
						eAux=error[(iter-1)*offset+globalTidX];
						(!isnan(eAux))?e+=eAux*0.4375:e+=0.0f;		
						if((globalTidX-1)>-1){
							eAux=error[(iter-1)*offset+(globalTidX-1)];
							(!isnan(eAux))?e+=eAux*0.1875:e+=0.0f;		
						}
					}
				}
				if((iter-2)>-1){
					eAux=error[(iter-2)*offset+(globalTidX)];
						(!isnan(eAux))?e+=eAux*0.3125:e+=0.0f;		
				}
				if((iter-3)>-1){
					eAux=error[(iter-3)*offset+(globalTidX)];
					(!isnan(eAux))?e+=eAux*0.0625:e+=0.0f;		
				}	

			}
			value+=e;
			error[iter*offset+globalTidX]=value-round(value);
			output[iter*offset+globalTidX]=round(value);
		}
	}
}
