
#include <hip/hip_runtime.h>
__global__ void restoreImage(const float *orderedImage, float *restoredImage, int m, int n, int x, int y){
	float value=0.0;
	int label=0;
	int pbr=0;
	int posI=0;
	int posJ=0;
	for(int globalTidY=threadIdx.y+blockIdx.y*blockDim.y;globalTidY<y;globalTidY+=blockDim.y*gridDim.y){
		for(int globalTidX=threadIdx.x+blockIdx.x*blockDim.x;globalTidX<x;globalTidX+=blockDim.x*gridDim.x){
			value=orderedImage[x*globalTidY+globalTidX];

			if(!isnan(value)){
				label=globalTidY+1;
				pbr=(label<=n)?0: (label-n)/2 + (label-n)%2;
				posI=globalTidX+pbr;
				posJ=label-1-2*posI;
				restoredImage[posI*n+posJ]=value;
			}			
		}
	}
}
