
#include <hip/hip_runtime.h>
__global__ void reorderImage(const float *originalImage, float *orderedImage, int m, int n){
	
	float value=0.0;
	int label=0;
	int pbr=0;
	int posInGroup=0;
	for(int globalTidY=threadIdx.y+blockIdx.y*blockDim.y;globalTidY<m;globalTidY+=blockDim.y*gridDim.y){
		for(int globalTidX=threadIdx.x+blockIdx.x*blockDim.x;globalTidX<n;globalTidX+=blockDim.x*gridDim.x){
			value=originalImage[n*globalTidY+globalTidX];
			label=globalTidX+2*globalTidY+1;
			pbr=(label<=n)?0: (label-n)/2 + (label-n)%2;
		
			posInGroup=globalTidY-pbr;
			
			
			orderedImage[min(m,n/2 + n%2)*(label-1)+posInGroup]=value;//globalTidX+2*globalTidY;
			
		}
	}

}
